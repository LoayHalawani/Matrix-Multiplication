#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define M 300
#define N 500
#define K 200
#define TILE 16

__host__ void generate(int *matrix, int rows, int cols) {
    int i, j;
    for(i = 0; i < rows; i ++) {
        for(j = 0; j < cols; j ++) {
            matrix[i * cols + j] = rand() % 900 + 100;
        }
    }
}

/*
__host__ void display(int *matrix, int rows, int cols) {
    for(int i = 0; i < rows; i ++) {
        for(int j = 0; j < cols; j ++) {
            printf("%d ", matrix[i * cols + j]);
        }
        printf("\n");
    }
}
*/

__global__ void mat_mul(int *A, int *B, int *C) {
    __shared__ int shared_a[TILE][TILE];
    __shared__ int shared_b[TILE][TILE];

    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;

    int rows = by * TILE + ty;
    int cols = bx * TILE + tx;

    int i, Cvalue = 0;
    for(i = 0; i < (N - 1) / (TILE + 1); i ++) {
        if(rows < M && (i * TILE + tx) < N) {
            shared_a[ty][tx] = A[rows * N + i * TILE + tx];
        }
        else {
            shared_a[ty][tx] = 0;
        }

        if(i * TILE + ty < N && cols < K) {
            shared_b[ty][tx] = B[(i * TILE + ty) * K + cols];
        }
        else {
            shared_b[ty][tx] = 0;
        }

        __syncthreads();

        for(int j = 0; j < TILE; j ++) {
            Cvalue += shared_a[ty][j] * shared_b[j][tx];
        }

        __syncthreads();
    }

    if(rows < M && cols < K) {
        C[rows * K + cols] = Cvalue;
    }
}

int main() {
    int *h_a, *h_b, *h_c;
    int *d_a, *d_b, *d_c;
    
    int s_a = M * N * sizeof(int);
    int s_b = N * K * sizeof(int);
    int s_c = M * K * sizeof(int);

    h_a = (int *)malloc(s_a);
    h_b = (int *)malloc(s_b);
    h_c = (int *)malloc(s_c);

    srand(time(NULL));
    generate(h_a, M, N);
    generate(h_b, N, K);

    hipMalloc((void **)&d_a, s_a);
    hipMalloc((void **)&d_b, s_b);
    hipMalloc((void **)&d_c, s_c);

    hipMemcpy(d_a, h_a, s_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, s_b, hipMemcpyHostToDevice);

    dim3 DimBlock(TILE, TILE, 1);
    dim3 DimGrid((K + DimBlock.x - 1) / DimBlock.x, (M + DimBlock.y - 1) / DimBlock.y, 1);

    printf("Multiplying random matrices A(300x500) and B(500x200)...\n");

    clock_t start = clock();

    mat_mul<<<DimGrid, DimBlock>>>(d_a, d_b, d_c);

    clock_t end = clock();

    hipMemcpy(h_c, d_c, s_c, hipMemcpyDeviceToHost);

    /*
    printf("Resulting Matrix C(300x200):\n");
    display(h_c, M, K);
    */

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(h_a);
    free(h_b);
    free(h_c);

    double time = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("Time taken: %f ms", time * 1000);

    return 0;
}
