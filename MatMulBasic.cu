#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define M 300
#define N 500
#define K 200

__host__ void generate(int *matrix, int rows, int cols) {
    int i, j;
    for(i = 0; i < rows; i ++) {
        for(j = 0; j < cols; j ++) {
            matrix[i * cols + j] = rand() % 900 + 100;
        }
    }
}

/*__host__ void display(int *matrix, int rows, int cols) {
    for(int i = 0; i < rows; i ++) {
        for(int j = 0; j < cols; j ++) {
            printf("%d ", matrix[i * cols + j]);
        }
        printf("\n");
    }
}*/

__global__ void mat_mul(int *A, int *B, int *C) {
    int row = (blockIdx.y * blockDim.y) + threadIdx.y;
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(row < M && col < K) {
        int i, sum = 0;
        for(i = 0; i < N; i ++) {
            sum += A[row * N + i] * B[i * K + col];
        }
        C[row * K + col] = sum;
    }
}

int main() {
    int *h_a, *h_b, *h_c;
    int *d_a, *d_b, *d_c;
    
    int s_a = M * N * sizeof(int);
    int s_b = N * K * sizeof(int);
    int s_c = M * K * sizeof(int);

    h_a = (int *)malloc(s_a);
    h_b = (int *)malloc(s_b);
    h_c = (int *)malloc(s_c);

    srand(time(NULL));
    generate(h_a, M, N);
    generate(h_b, N, K);

    hipMalloc((void **)&d_a, s_a);
    hipMalloc((void **)&d_b, s_b);
    hipMalloc((void **)&d_c, s_c);

    hipMemcpy(d_a, h_a, s_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, s_b, hipMemcpyHostToDevice);

    dim3 DimBlock(16, 16, 1);
    dim3 DimGrid((K + DimBlock.x - 1) / DimBlock.x, (M + DimBlock.y - 1) / DimBlock.y, 1);

    printf("Multiplying random matrices A (300x500) and B (500x200)...\n");

    clock_t start = clock();

    mat_mul<<<DimGrid, DimBlock>>>(d_a, d_b, d_c);

    clock_t end = clock();

    hipMemcpy(h_c, d_c, s_c, hipMemcpyDeviceToHost);

    //printf("Resulting Matrix C (300x200):\n");
    //display(h_c, M, K);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(h_a);
    free(h_b);
    free(h_c);

    double time = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("Time taken: %f ms", time * 1000);

    return 0;
}
